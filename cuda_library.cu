#include "hip/hip_runtime.h"
//
// Created by Daniel Simon on 7/18/19.
//

#include "cuda_library.cuh"

namespace cuda {

__global__ void Kernel_cu(const float* x, float* y, int n, float a, float b) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  y[idx] = a * x[idx] + b;
}

hipError_t Kernel(const float* x, float* y, int n, float a, float b) {
  float* x_cu;
  float* y_cu;
  hipMalloc(&x_cu, sizeof(float) * n);
  hipMemcpy(x_cu, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMalloc(&y_cu, sizeof(float) * n);
  Kernel_cu<<<n / 128, 128>>>(x_cu, y_cu, n, a, b);
  hipMemcpy(y, y_cu, sizeof(float) * n, hipMemcpyDeviceToHost);
  return hipGetLastError();
}

}
